#include "hip/hip_runtime.h"
// chain/chain-kernels.cu

// Copyright  2015  Johns Hopkins University (author: Daniel Povey)


// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.


#include <cfloat>
#include "chain/chain-kernels-ansi.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 200
#error - Kaldi no longer supports CC1.x devices. Please use a newer GPU or \
         configure with --use-cuda=no (this will disable the use of GPU).
#endif


#ifdef __HIPCC__
#if ( __CUDACC_VER_MAJOR__ >= 8 ) && ( !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600 )
// native implementation available
#else
#if __CUDA_ARCH__ >= 600
#error using CAS implementation of double atomicAdd
#endif
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif
#endif


template <typename Real>
__device__ inline void atomic_add(Real* address, Real value) {
  atomicAdd(address, value);
}

template <typename Real>
__device__ inline void atomic_add_thresholded(Real* address, Real value) {
  // This function uses a randomized algorithm to only do atomic adds for values
  // with absolute value >= a threshold, 
  // and if it's below the threshold, randomly add the
  // threshold itself with probability (value / threshold).  This preserves
  // expectations.  

  // kThresholdingPowerOfTwo is defined in chain-datastruct.h; it defines
  // the threshold for randomized posterior pruning.
  const Real threshold = 1.0 / (1 << kThresholdingPowerOfTwo);
  Real abs_value = abs(value);
  if (abs_value >= threshold) {
    atomic_add(address, value);
  } else {
    // The intention here is to do:
    // with probability(value / threshold), do:
    //   atomic_add(address, threshold);
    // We use the least significant bits of the value as a source of
    // randomness.  It would probably be more efficient to extract these
    // random bits directly from the float, but I don't want to have to
    // deal with endian-ness issues.
    //
    // below, x is a fixed-point representation of (value / threshold); it would
    // be 16777216 == 2^24 if value == threshold and 0 if value == 0.  We choose
    // the power 24 because that's the number of binary digits in the mantissa
    // in IEEE single precision floating point.
    // Note: we parenthesize the expression like this so that the
    // denominator can be precomputed as a constant expression.
    int32_cuda x = abs_value / (threshold / (1 << 24));
    // in the line below, the expression (x >> 12) is a representation of (value /
    // threshold) between 0 and 4096, with 4096 representing (value / threshold ==
    // 1), while (x & 4095) is treated as a pseudorandom number between 0 and 4095.
    if ((x >> 12) > (x & 4095)) {
      if (value >= 0) atomic_add(address, threshold);
      else atomic_add(address, -threshold);
    }
  }
}

// one iteration of the forward computation in the chain HMM with 
// SMBR objective.
// The grid y determines which HMM-state we handle.  [put this in the grid because
// HMM-states don't all take the same amount of time in the backwards direction, and it's
// better for scheduling to have them at the outer level.]
// The block x and grid x determine which sequence (0 ... num_sequences - 1) we handle;
// note that num_sequences == the number of elements in the minibatch, and we
// insist they all have the same number of time steps.
// note: 'probs' is indexed by sequence-index + (pdf-index * prob_stride).
// note: 'num_post' is indexed by sequence-index + (pdf-index * post_stride).
__global__
static void _cuda_chain_smbr_hmm_forward(
    const Int32Pair *backward_transitions,
    const DenominatorGraphTransition *transitions,
    int32_cuda num_sequences,
    int32_cuda num_hmm_states,
    const BaseFloat *probs, int32_cuda prob_stride, 
    const BaseFloat *num_post, int32_cuda post_stride,
    const BaseFloat *prev_alpha, const BaseFloat *prev_alpha_smbr,
    BaseFloat *this_alpha, BaseFloat *this_alpha_smbr) {
  // 'backward_transitions', indexed by hmm-state, consists of [start, end]
  // indexes into the 'transitions' array.  This gives us the info for
  // transitions *into* this state.  'probs' contains the exponentiated neural
  // net outputs; it has dimension num-output-indexes by num_sequences and its
  // stride is 'prob_stride'.  'prev_alpha' and 'this_alpha', which are
  // extracted from a larger matrix, both have dimension num-history-states by
  // num-sequences. 'prev_alpha_smbr' and 'this_alpha_smbr' are analogous 
  // for the partial SMBR values.

  // s is the index of the sequence within the minibatch,
  // from 0 .. num-egs-in-this-minibatch - 1.
  // h is the hmm-state index.
  int32_cuda s = threadIdx.x + blockIdx.x * blockDim.x,
      h  = blockIdx.y;
  if (s >= num_sequences)
    return;

  double this_tot_alpha = 0.0, this_tot_alpha_smbr = 0.0;
  const DenominatorGraphTransition
      *trans_iter = transitions + backward_transitions[h].first,
      *trans_end = transitions + backward_transitions[h].second;
  // Note: regarding this loop unrolling, I tried the automatic unrolling using
  // #pragma unroll 2 (after modifying the loop to have an integer index), but I
  // did not see any performance improvement, it was slightly slower.  So the
  // compiler must be doing something different than what I'm doing here.
  const int loop_unroll = 2;  // don't change this without changing the code
                              // below.
  for (; trans_iter + loop_unroll <= trans_end; trans_iter += loop_unroll) {
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        prev_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat transition_prob1 = trans_iter[1].transition_prob;
    int32_cuda pdf_id1 = trans_iter[1].pdf_id,
        prev_hmm_state1 = trans_iter[1].hmm_state;
    BaseFloat pseudo_loglike0 = probs[pdf_id0 * prob_stride + s],
        num_post0 = num_post[pdf_id0 * post_stride + s],
        this_prev_alpha0 = prev_alpha[prev_hmm_state0 * num_sequences + s],
        this_prev_alpha_smbr0 = 
               prev_alpha_smbr[prev_hmm_state0 * num_sequences + s],
        pseudo_loglike1 = probs[pdf_id1 * prob_stride + s],
        num_post1 = num_post[pdf_id1 * post_stride + s],
        this_prev_alpha1 = prev_alpha[prev_hmm_state1 * num_sequences + s],
        this_prev_alpha_smbr1 =
          prev_alpha_smbr[prev_hmm_state1 * num_sequences + s];

    this_tot_alpha += this_prev_alpha0 * transition_prob0 * pseudo_loglike0 +
                       this_prev_alpha1 * transition_prob1 * pseudo_loglike1;
    this_tot_alpha_smbr += 
      (this_prev_alpha_smbr0 + num_post0) * this_prev_alpha0 
      * transition_prob0 * pseudo_loglike0
      + (this_prev_alpha_smbr1 + num_post1) * this_prev_alpha1 
      * transition_prob1 * pseudo_loglike1;
  }
  if (trans_iter != trans_end) {
    // mop up the odd transition.
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
       prev_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat pseudo_loglike0 = probs[pdf_id0 * prob_stride + s],
        num_post0 = num_post[pdf_id0 * post_stride + s],
        this_prev_alpha0 = prev_alpha[prev_hmm_state0 * num_sequences + s],
        this_prev_alpha_smbr0 = 
          prev_alpha_smbr[prev_hmm_state0 * num_sequences + s];
    this_tot_alpha += this_prev_alpha0 * transition_prob0 * pseudo_loglike0;
    this_tot_alpha_smbr += 
      (this_prev_alpha_smbr0 + num_post0) * this_prev_alpha0 
      * transition_prob0 * pseudo_loglike0;
  }

  // Let arbitrary_scale be the inverse of the sum of all alpha values on-- the
  // previous frame this sum of all the alpha values is stored in the place that
  // we'd store the previous alpha for state-index equal to num_hmm_states
  // (i.e. one past the end).  We multiply this into all the
  // transition-probabilities from the previous frame to this frame, in both the
  // forward and backward passes, in order to keep the alphas in a good numeric
  // range.  This won't affect the posteriors, as it's just a constant factor
  // for each frame, but when computing the total likelihood we'll need to
  // compensate for it later on.
  BaseFloat arbitrary_scale = 
      1.0 / prev_alpha[num_hmm_states * num_sequences + s];
  this_alpha[h * num_sequences + s] = this_tot_alpha * arbitrary_scale;
  if (this_tot_alpha > 0.0)
    this_alpha_smbr[h * num_sequences + s] = 
      this_tot_alpha_smbr / this_tot_alpha;
  else 
    this_alpha_smbr[h * num_sequences + s] = 0.0;
}


__global__
static void _cuda_chain_smbr_hmm_backward(
    const Int32Pair *forward_transitions,
    const DenominatorGraphTransition *transitions,
    int32_cuda num_sequences, int32_cuda num_hmm_states,
    const BaseFloat *probs, int32_cuda prob_stride, 
    const BaseFloat *num_post, int32_cuda post_stride,
    const BaseFloat *tot_smbr,
    const BaseFloat *this_alpha, const BaseFloat *this_alpha_smbr,
    const BaseFloat *next_beta, const BaseFloat *next_beta_smbr,
    BaseFloat *this_beta, BaseFloat *this_beta_smbr,
    BaseFloat *acc_deriv, int32_cuda acc_deriv_stride,
    BaseFloat *log_prob_deriv, int32_cuda log_prob_deriv_stride) {
  // 'forward_transitions', indexed by hmm-state, consists of [start, end]
  // indexes into the 'transition_info' array.  This is about the transitions
  // *out of* this state.  'probs' contains the exponentiated neural net
  // outputs; it has dimension num-output-indexes by num_sequences, and contains
  // just the observation probabilities for this time index.  Its stride is
  // prob_stride.
  // 'this_alpha', 'next_beta' and 'this_beta' all have dimension
  // num-history-states by num-sequences.
  // 'this_alpha_smbr', 'next_beta_smbr', and 'this_beta_smbr' are 
  // analogous quantities storing values for SMBR objective.
  // The beta probs are normalized in such a way (by multiplying by 1/(total-data-prob))
  // that to get occupation counts we don't need to multiply by 1/total-data-prob.
  // deriv_scale is a factor (e.g. -1.0 or -0.99) that we multiply these derivs by
  // while accumulating them.

  // s is the index of the sequence within the minibatch,
  // from 0 .. num-egs-in-this-minibatch - 1.
  // h is the hmm-state index.
  int32_cuda s = threadIdx.x + blockIdx.x * blockDim.x,
      h = blockIdx.y;
  if (s >= num_sequences)
    return;

  // See where arbitrary_scale is defined in the forward computation above, for
  // more explanation of inv_arbitrary_scale.
  BaseFloat this_alpha_prob = this_alpha[h * num_sequences + s],
      this_alpha_smbr_i = this_alpha_smbr[h * num_sequences + s],
      inv_arbitrary_scale =
      this_alpha[num_hmm_states * num_sequences + s];
  double tot_variable_factor = 0.0, tot_beta_smbr = 0.0;

  BaseFloat occupation_factor = this_alpha_prob / inv_arbitrary_scale;
  const DenominatorGraphTransition
      *trans_iter = transitions + forward_transitions[h].first,
      *trans_end = transitions + forward_transitions[h].second;
  const int loop_unroll = 2;  // don't change this without changing the code
                              // below.
  for (; trans_iter + loop_unroll <= trans_end; trans_iter += loop_unroll) {
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        next_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat transition_prob1 = trans_iter[1].transition_prob;
    int32_cuda pdf_id1 = trans_iter[1].pdf_id,
        next_hmm_state1 = trans_iter[1].hmm_state;
    BaseFloat next_beta_j0 = next_beta[next_hmm_state0 * num_sequences + s],
        next_beta_smbr_j0 = next_beta_smbr[next_hmm_state0 * num_sequences + s],
        next_beta_j1 = next_beta[next_hmm_state1 * num_sequences + s],
        next_beta_smbr_j1 = next_beta_smbr[next_hmm_state1 * num_sequences + s],
        prob0 = probs[pdf_id0 * prob_stride + s],
        prob1 = probs[pdf_id1 * prob_stride + s],
        num_post0 = num_post[pdf_id0 * post_stride + s], 
        num_post1 = num_post[pdf_id1 * post_stride + s];

    BaseFloat variable_factor0 = transition_prob0 * next_beta_j0 * prob0,
        variable_factor1 = transition_prob1 * next_beta_j1 * prob1;
    tot_beta_smbr += (next_beta_smbr_j0 + num_post0) * variable_factor0
      + (next_beta_smbr_j1 + num_post1) * variable_factor1;
    tot_variable_factor += variable_factor0 + variable_factor1;
    BaseFloat occupation_prob0 = variable_factor0 * occupation_factor;
    BaseFloat this_acc_r0 = occupation_prob0 
      * (this_alpha_smbr_i + num_post0 + next_beta_smbr_j0);
    atomic_add(acc_deriv + (pdf_id0 * acc_deriv_stride + s),
               this_acc_r0);
    atomic_add(log_prob_deriv + (pdf_id0 * log_prob_deriv_stride + s),
               occupation_prob0);
    BaseFloat occupation_prob1 = variable_factor1 * occupation_factor;
    BaseFloat this_acc_r1 = occupation_prob1
      * (this_alpha_smbr_i + num_post1 + next_beta_smbr_j1);
    atomic_add(acc_deriv + (pdf_id1 * acc_deriv_stride + s),
               this_acc_r1);
    atomic_add(log_prob_deriv + (pdf_id1 * log_prob_deriv_stride + s),
               occupation_prob1);
  }
  if (trans_iter != trans_end) {
    // mop up the odd transition.
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        next_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat next_beta_j0 = next_beta[next_hmm_state0 * num_sequences + s],
        next_beta_smbr_j0 = next_beta_smbr[next_hmm_state0 * num_sequences + s],
        prob0 = probs[pdf_id0 * prob_stride + s],
        num_post0 = num_post[pdf_id0 * post_stride + s];
    BaseFloat variable_factor0 = transition_prob0 * next_beta_j0 * prob0;
    tot_beta_smbr += (next_beta_smbr_j0 + num_post0) * variable_factor0;
    tot_variable_factor += variable_factor0;
    BaseFloat occupation_prob0 = variable_factor0 * occupation_factor;
    BaseFloat this_acc_r0 = occupation_prob0
      * (this_alpha_smbr_i + num_post0 + next_beta_smbr_j0);
    atomic_add(acc_deriv + (pdf_id0 * acc_deriv_stride + s),
               this_acc_r0);
    atomic_add(log_prob_deriv + (pdf_id0 * log_prob_deriv_stride + s),
               occupation_prob0);
  }
  BaseFloat beta = tot_variable_factor / inv_arbitrary_scale;
  this_beta[h * num_sequences + s] = beta;
  if (tot_variable_factor > 0.0)
    this_beta_smbr[h * num_sequences + s] = 
        tot_beta_smbr / tot_variable_factor;
  else
    this_beta_smbr[h * num_sequences + s] = 0.0;
}


// Chain forward with SMBR objective
void cuda_chain_smbr_hmm_forward(
    dim3 Gr, dim3 Bl,
    const Int32Pair *backward_transitions,
    const DenominatorGraphTransition *transitions,
    int32_cuda num_sequences,
    int32_cuda num_hmm_states,
    const BaseFloat *probs, int32_cuda prob_stride,
    const BaseFloat *num_post, int32_cuda post_stride,
    const BaseFloat *prev_alpha, const BaseFloat *prev_alpha_smbr,
    BaseFloat *this_alpha, BaseFloat *this_alpha_smbr) {
  _cuda_chain_smbr_hmm_forward<<<Gr,Bl>>>(
      backward_transitions, transitions,
      num_sequences, num_hmm_states,
      probs, prob_stride, num_post, post_stride,
      prev_alpha, prev_alpha_smbr, this_alpha, this_alpha_smbr);
}

void cuda_chain_smbr_hmm_backward(
    dim3 Gr, dim3 Bl,
    const Int32Pair *forward_transitions,
    const DenominatorGraphTransition *transitions,
    int32_cuda num_sequences,
    int32_cuda num_hmm_states,
    const BaseFloat *probs, int32_cuda prob_stride,
    const BaseFloat *num_post, int32_cuda post_stride,
    const BaseFloat *tot_smbr,
    const BaseFloat *this_alpha, const BaseFloat *this_alpha_smbr,
    const BaseFloat *next_beta, const BaseFloat *next_beta_smbr,
    BaseFloat *this_beta, BaseFloat *this_beta_smbr,
    BaseFloat *acc_deriv,
    int32_cuda acc_deriv_stride,
    BaseFloat *log_prob_deriv,
    int32_cuda log_prob_deriv_stride) {
  _cuda_chain_smbr_hmm_backward<<<Gr,Bl>>>(
      forward_transitions, transitions,
      num_sequences, num_hmm_states,
      probs, prob_stride, num_post, post_stride, tot_smbr,
      this_alpha, this_alpha_smbr, next_beta, next_beta_smbr,
      this_beta, this_beta_smbr, 
      acc_deriv, acc_deriv_stride, 
      log_prob_deriv, log_prob_deriv_stride);
}
